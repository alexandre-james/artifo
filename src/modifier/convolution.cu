#include "hip/hip_runtime.h"
#include "convolution.hpp"
#include "../image/gray_image.hpp"
#include "../image/rgb_image.hpp"
#include "../image/rgba_image.hpp"
#include "../image/hsv_image.hpp"

#include <cstdio>

#define gpu_check(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ uint8_t device_bound(int nb) {
    if (nb > MAX_LEVEL)
        return MAX_LEVEL;
    if (nb < 0)
        return 0;
    return nb;
}

__global__ void convolve_pixel(uint8_t *output, uint8_t *input, float *kernel,
    int input_width, int input_height, int kernel_width, int kernel_height) {

    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (y >= input_height || x >= input_width)
        return;

    float sum = 0;

    for (int i = 0; i < kernel_height; i++) {
        for (int j = 0; j < kernel_width; j++) {
            int y_pos = y - kernel_height / 2 + i;
            int x_pos = x - kernel_width / 2 + j;
            uint8_t value = 0;
            if (x_pos >= 0 && y_pos >= 0 && y_pos < input_height && x_pos < input_width)
                value = input[y_pos * input_width + x_pos];
            sum += (float) kernel[i * kernel_width + j] * value;
        }
    }
    output[y * input_width + x] = device_bound(sum);
}

template <typename image_type>
image_type *convolve(image_type *input, mask *kernel) {
    int tx = 24;
    int ty = 16;

    dim3 blocks(input->width * input->dim / tx + 1, input->height / ty + 1);
    dim3 threads(tx, ty);

    uint8_t *output_gpu;
    uint8_t *input_gpu;
    float *kernel_gpu;

    hipMalloc((void **) &output_gpu, input->length * sizeof(uint8_t));
    gpu_check(hipGetLastError());

    hipMalloc((void **) &input_gpu, input->length * sizeof(uint8_t));
    gpu_check(hipGetLastError());

    hipMalloc((void **) &kernel_gpu, kernel->width * kernel->height * sizeof(float));
    gpu_check(hipGetLastError());

    hipMemcpy(input_gpu, input->pixels, input->length * sizeof(uint8_t), hipMemcpyHostToDevice);
    gpu_check(hipGetLastError());

    hipMemcpy(kernel_gpu, kernel->values, kernel->width * kernel->height * sizeof(float), hipMemcpyHostToDevice);
    gpu_check(hipGetLastError());
    
    convolve_pixel<<<blocks, threads>>>(output_gpu, input_gpu, kernel_gpu,
    input->width * input->dim, input->height, kernel->width, kernel->height);
    gpu_check(hipGetLastError());
    gpu_check(hipDeviceSynchronize());

    image_type *output = new image_type(input->width, input->height);
    hipMemcpy(output->pixels, output_gpu, output->length * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipFree(output_gpu);
    hipFree(input_gpu);
    hipFree(kernel_gpu);

    return output;
}

template gray_image *convolve(gray_image *, mask *);
template rgb_image *convolve(rgb_image *, mask *);
template rgba_image *convolve(rgba_image *, mask *);
template hsv_image *convolve(hsv_image *, mask *);